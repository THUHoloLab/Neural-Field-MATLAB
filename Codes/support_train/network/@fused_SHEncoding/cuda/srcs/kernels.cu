#include "hip/hip_runtime.h"
#include "kernels.cuh"
#include <hip/hip_runtime.h>

__global__ void SH_Encoding_forward_kernel(
    const float3 * __restrict__ dirs,
    const dim3 dirs_sz,
    float * __restrict__ embedding
){
    unsigned batch_idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (batch_idx >= dirs_sz.x) return;
    float x = dirs[batch_idx].x;
    float y = dirs[batch_idx].y;
    float z = dirs[batch_idx].z;
    float xy = x * y;
    float xz = x * z;
    float yz = z * y;
    float x2 = x * x;
    float y2 = y * y;
    float z2 = z * z;
    // l = 0

    unsigned bathc_idx16 = batch_idx * 16;

    embedding[bathc_idx16 + 0] = (0.28209479177387814f);
    // l = 1
    embedding[bathc_idx16 + 1] = (-0.48860251190291987f * y);
    embedding[bathc_idx16 + 2] = (0.48860251190291987f * z);
    embedding[bathc_idx16 + 3] = (-0.48860251190291987f * x);
    // l = 2
    embedding[bathc_idx16 + 4] = (1.0925484305920792f * xy);
    embedding[bathc_idx16 + 5] = (-1.0925484305920792f * yz);
    embedding[bathc_idx16 + 6] = (0.94617469575755997f * z2 - 0.31539156525251999f);
    embedding[bathc_idx16 + 7] = (-1.0925484305920792f * xz);
    embedding[bathc_idx16 + 8] = (0.54627421529603959f * x2 - 0.54627421529603959f * y2);
    // l = 3
    embedding[bathc_idx16 + 9] = (0.59004358992664352f * y * (-3.0 * x2 + y2));
    embedding[bathc_idx16 + 10] = (2.8906114426405538f * xy * z);
    embedding[bathc_idx16 + 11] = (0.45704579946446572f * y * (1.0 - 5.0 * z2));
    embedding[bathc_idx16 + 12] = (0.3731763325901154f * z * (5.0 * z2 - 3.0));
    embedding[bathc_idx16 + 13] = (0.45704579946446572f * x * (1.0 - 5.0 * z2));
    embedding[bathc_idx16 + 14] = (1.4453057213202769f * z * (x2 - y2));
    embedding[bathc_idx16 + 15] = (0.59004358992664352f * x * (-x2 + 3.0 * y2));
}

__global__ void SH_Encoding_backward_kernel(
    const float * __restrict__ dl_dembedding,  // dL/dembedding, shape [16,batch_size]
    const float3 * __restrict__ dirs,           // input direction
    const dim3 dirs_sz,                         // batch size
    float3 * __restrict__ dl_ddirs             // output dL/ddirs, shape [batch_size]
){
    unsigned batch_idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (batch_idx >= dirs_sz.x) return;

    float x = dirs[batch_idx].x;
    float y = dirs[batch_idx].y;
    float z = dirs[batch_idx].z;

    // collection current batch's gradient 16 channels
    float g[16];
    #pragma unroll
    for (int i = 0; i < 16; ++i) {
        g[i] = dl_dembedding[batch_idx * 16 + i];
    }

    // preloading intermediate variables
    float xy = x * y;
    float xz = x * z;
    float yz = y * z;
    float x2 = x * x;
    float y2 = y * y;
    float z2 = z * z;

    // for dL/dx, dL/dy, dL/dz
    float dL_dx = 0.0f;
    float dL_dy = 0.0f;
    float dL_dz = 0.0f;

    // l=0: 
    // l=1
    dL_dx += -0.48860251190291987f * g[3];  // d(embedding[3])/dx = -0.488...
    dL_dy += -0.48860251190291987f * g[1];  // d(embedding[1])/dy = -0.488...
    dL_dz += 0.48860251190291987f * g[2];   // d(embedding[2])/dz = 0.488...

    // l=2
    dL_dx += 1.0925484305920792f * y * g[4] - 1.0925484305920792f * z * g[7] + 2 * 0.54627421529603959f * x * g[8];
    dL_dy += 1.0925484305920792f * x * g[4] - 1.0925484305920792f * z * g[5] - 2 * 0.54627421529603959f * y * g[8];
    dL_dz += -1.0925484305920792f * y * g[5] + (0.94617469575755997f * 2 * z) * g[6] - 1.0925484305920792f * x * g[7];

    // l=3
    dL_dx += 0.59004358992664352f * y * (-6.0f * x) * g[9] + 
             2.8906114426405538f * y * z * g[10] + 
             0.45704579946446572f * (1.0f - 5.0f * z2) * g[13] + 
             1.4453057213202769f * z * 2 * x * g[14] + 
             0.59004358992664352f * (-3.0f * x2 + 3.0f * y2) * g[15];

    dL_dy += 0.59004358992664352f * (-3.0f * x2 + y2 + y * 2 * y) * g[9] + 
             2.8906114426405538f * x * z * g[10] + 
             0.45704579946446572f * (1.0f - 5.0f * z2) * g[11] - 
             1.4453057213202769f * z * 2 * y * g[14] + 
             0.59004358992664352f * x * 6.0f * y * g[15];

    dL_dz += 2.8906114426405538f * xy * g[10] + 
             0.45704579946446572f * y * (-10.0f * z) * g[11] + 
             0.3731763325901154f * (15.0f * z2 - 3.0f) * g[12] + 
             0.45704579946446572f * x * (-10.0f * z) * g[13] + 
             1.4453057213202769f * (x2 - y2) * g[14];

    // gradient
    dl_ddirs[batch_idx] = make_float3(dL_dx, dL_dy, dL_dz);
}