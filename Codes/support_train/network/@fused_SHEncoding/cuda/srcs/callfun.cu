#include "hip/hip_runtime.h"
#include "addon/helpers.cuh"
#include "callfun.cuh"

__host__ dim3 size2dim3( const mxGPUArray * in){
    const mwSize *sz = mxGPUGetDimensions(in);
    const int  dim = (int) mxGPUGetNumberOfDimensions(in);
    dim3 imgSz;
    imgSz = {(unsigned) sz[0], (unsigned) sz[1], 1};
    if (dim > 2){
        imgSz.z = (unsigned) sz[2];
    }
    return imgSz;
}

void Call_Fwd(
    int nlhs, mxArray *plhs[], 
    int nrhs, mxArray const * prhs[]
){
    const mxGPUArray * dir_batch;
    mxGPUArray * output_embedding;

    mxInitGPU();
    dir_batch = mxGPUCreateFromMxArray(prhs[0]);

    const int feature_dim = (const int) mxGetPr(prhs[1])[0];
    const dim3 dir_sz = size2dim3(dir_batch);
    const float3 *d_dir_batch = (const float3 *) mxGPUGetDataReadOnly(dir_batch);

    CHECK_THROW((size_t) dir_sz.x == 3);
    CHECK_THROW(feature_dim == 4);
    // mexPrintf("dir_batch size is %u %u %u \n", dir_sz.x, dir_sz.y, dir_sz.z);
    dim3 N_BLOCKS = {
        (unsigned) (dir_sz.y + N_THREADS - 1) / N_THREADS,
        1
    };

    const mwSize sz[2] = {16, dir_sz.y};
    dim3 output_sz = {(unsigned) sz[0], (unsigned) sz[1], 1};
    output_embedding = mxGPUCreateGPUArray(
        mxGPUGetNumberOfDimensions(dir_batch),
        sz,
        mxSINGLE_CLASS,
        mxREAL,
        MX_GPU_INITIALIZE_VALUES
    );  

    float * d_output_embedding = (float *) mxGPUGetData(output_embedding);

    SH_Encoding_forward_kernel<<<N_BLOCKS, N_THREADS>>>(
        d_dir_batch,
        dir_sz,
        /// output
        d_output_embedding
    );

    plhs[0] = mxGPUCreateMxArrayOnGPU(output_embedding);

    mxGPUDestroyGPUArray(dir_batch);
    mxGPUDestroyGPUArray(output_embedding);
}

void Call_Bwd(
    int nlhs, mxArray *plhs[], 
    int nrhs, mxArray const * prhs[]
){
    const mxGPUArray * dir_batch;
    const mxGPUArray * dl_doutput;

    mxGPUArray * dl_ddir;

    mxInitGPU();
    dir_batch   = mxGPUCreateFromMxArray(prhs[0]);
    dl_doutput  = mxGPUCreateFromMxArray(prhs[1]);

    int feature_dim = (int) mxGetPr(prhs[2])[0];


    dim3 dir_sz = size2dim3(dir_batch);
    dim3 dl_doutput_sz = size2dim3(dl_doutput);
    // mexPrintf("dir_batch size is %u %u %u \n", dir_sz.x, dir_sz.y, dir_sz.z);
    // mexPrintf("dl_doutput_sz size is %u %u %u \n", dl_doutput_sz.x, dl_doutput_sz.y, dl_doutput_sz.z);
    CHECK_THROW((size_t) dir_sz.x == 3);
    CHECK_THROW((size_t) dir_sz.y == (size_t) dl_doutput_sz.y);
    CHECK_THROW((size_t) dl_doutput_sz.x == (size_t) 16);

    const float3 * d_xyz_batch = (const float3 *) mxGPUGetDataReadOnly(dir_batch);
    const float * d_dl_doutput = (const float *) mxGPUGetDataReadOnly(dl_doutput);

    dim3 N_BLOCKS = {
        (unsigned) (dir_sz.y + N_THREADS - 1) / N_THREADS,
        1
    };
    // gradient of embedding
    dl_ddir = mxGPUCreateGPUArray(
        mxGPUGetNumberOfDimensions(dir_batch),
        mxGPUGetDimensions(dir_batch),
        mxSINGLE_CLASS,
        mxREAL,
        MX_GPU_INITIALIZE_VALUES
    );  

    float3 * d_dl_ddir = (float3 *) mxGPUGetData(dl_ddir);

    SH_Encoding_backward_kernel<<<N_BLOCKS, N_THREADS>>>(
        d_dl_doutput,
        d_xyz_batch,
        dir_sz,
        /// output
        d_dl_ddir
    );

    plhs[0] = mxGPUCreateMxArrayOnGPU(dl_ddir);
    mxGPUDestroyGPUArray(dir_batch);
    mxGPUDestroyGPUArray(dl_doutput);
    mxGPUDestroyGPUArray(dl_ddir);
}