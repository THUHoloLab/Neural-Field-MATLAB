#include "hip/hip_runtime.h"
#include "kernels.cuh"
#include <hip/hip_runtime.h>
#include <hip/hip_cooperative_groups.h>

#define POS_DIM 3

__global__ void hashEncoding_Fwd_kernel(
    const float3 * __restrict__ xys,
    const float4 * __restrict__ embedding,
    const float * __restrict__ bbox,
    const uint32_t * __restrict__ hash_offsets,
    const uint32_t * __restrict__ hash_map_sizes,
    const dim3 output_sz,
    const float log_scale, // (log(res_high) - log(res_low)) / (level - 1)
    const float base_res,
    const uint32_t feature_dim,
    // outputs
    float4 * __restrict__ output_embedding
){
    const uint32_t batch_id = blockIdx.x * blockDim.x + threadIdx.x;
    const uint32_t level_id = blockIdx.y; // page id

    if (batch_id >= output_sz.y) {
        return;
    }

    float scale = grid_resolution((int) level_id, log_scale, base_res);
    uint32_t resolution = uint32_t(ceilf(scale)) + 1;
    float pos[POS_DIM] = {xys[batch_id].x, xys[batch_id].y, xys[batch_id].z};
    uint32_t pos_grid[POS_DIM];

    #pragma unroll
    for(uint32_t idx = 0; idx < POS_DIM; ++ idx){
        float bbox_min = bbox[idx * 2 + 0];
        float normalized_pos = (pos[idx] - bbox_min) / 
                               (bbox[idx * 2 + 1] - bbox_min) * scale;
        // normalized_pos = max(min(normalized_pos,1.0f),0.0f);

        float temp_pos = __floorf(normalized_pos);
        pos_grid[idx] = (uint32_t) temp_pos;
        pos[idx] = normalized_pos - temp_pos;
    }

    const uint32_t map_size = __ldg(&hash_map_sizes[level_id]);
    const uint32_t offsets  = __ldg(&hash_offsets[level_id]);

    float4 features = make_float4(0.0f, 0.0f, 0.0f, 0.0f);

    #pragma unroll
    for(uint32_t idx = 0; idx < 8; ++idx){
        float w = 1.0f;
        uint32_t loc_pos[POS_DIM];
        #pragma unroll
        for(uint32_t dim = 0; dim < POS_DIM; ++dim){
            if ((idx & (1 << dim)) == 0){
                loc_pos[dim] = pos_grid[dim];
                w *= 1 - pos[dim];
            }else{
                loc_pos[dim] = pos_grid[dim] + 1;
                w *= pos[dim];
            }
        }

        uint32_t hash_idx = grid_index(
            map_size, resolution,
            loc_pos[0], loc_pos[1], loc_pos[2]
        );
        // uint32_t table_index = (offsets + hash_idx) * feature_dim;
        // features.x = fmaf(w, __ldg(&embedding[table_index + 0]), features.x);
        // features.y = fmaf(w, __ldg(&embedding[table_index + 1]), features.y);
        // features.z = fmaf(w, __ldg(&embedding[table_index + 2]), features.z);
        // features.w = fmaf(w, __ldg(&embedding[table_index + 3]), features.w);
        float4 this_embedding = __ldg(&embedding[offsets + hash_idx]);
        features.x = fmaf(w, this_embedding.x, features.x);
        features.y = fmaf(w, this_embedding.y, features.y);
        features.z = fmaf(w, this_embedding.z, features.z);
        features.w = fmaf(w, this_embedding.w, features.w);
    }

    output_embedding[batch_id * (output_sz.x / 4) + level_id] = features;
}

__global__ void hashEncoding_Bwd_kernel(
    const float3 * __restrict__ xys,
    const float4 * __restrict__ dl_doutput,
    const float * __restrict__ bbox,
    const uint32_t * __restrict__ hash_offsets,
    const uint32_t * __restrict__ hash_map_sizes,
    const dim3 output_sz,
    const float log_scale, // (log(res_high) - log(res_low)) / (level - 1)
    const float base_res,
    const uint32_t feature_dim,
    float * __restrict__ dl_dembedding
){
    const uint32_t batch_id = blockIdx.x * blockDim.x + threadIdx.x;
    const uint32_t level_id = blockIdx.y; // page id

    if (batch_id >= output_sz.y) {
        return;
    }

    float scale = grid_resolution((int) level_id, log_scale, base_res);
    uint32_t resolution = (uint32_t) ceilf(scale) + 1;
    float pos[POS_DIM] = {xys[batch_id].x, xys[batch_id].y, xys[batch_id].z};

    uint32_t pos_grid[POS_DIM];
    #pragma unroll
    for(uint32_t idx = 0; idx < POS_DIM; ++ idx){
        float bbox_min = bbox[idx * 2 + 0];
        float normalized_pos = (pos[idx] - bbox_min) / 
                               (bbox[idx * 2 + 1] - bbox_min) * scale;
        // normalized_pos = max(min(normalized_pos,1.0f),0.0f);
        float temp_pos = floorf(normalized_pos);
        pos_grid[idx] = (uint32_t) temp_pos;
        pos[idx] = normalized_pos - temp_pos;
    }

    const uint32_t map_size = __ldg(&hash_map_sizes[level_id]);
    const uint32_t offsets  = __ldg(&hash_offsets[level_id]);

    float4 dl_outputs = __ldg(&dl_doutput[batch_id * (output_sz.x / 4) + level_id]);

    #pragma unroll
    for(uint32_t idx = 0; idx < 8; ++idx){
        float w = 1.0f;
        uint32_t loc_pos[POS_DIM];
        #pragma unroll
        for(uint32_t dim = 0; dim < POS_DIM; ++dim){
            if ((idx & (1 << dim)) == 0){
                loc_pos[dim] = pos_grid[dim];
                w *= 1 - pos[dim];
            }else{
                loc_pos[dim] = pos_grid[dim] + 1;
                w *= pos[dim];
            }
        }

        uint32_t hash_idx = grid_index(
            map_size, resolution,
            loc_pos[0], loc_pos[1], loc_pos[2]
        );

        uint32_t base_id = (offsets + hash_idx) * feature_dim;
        float *temp = (float *) dl_dembedding;
        atomicAdd(temp + base_id + 0, w * dl_outputs.x);
        atomicAdd(temp + base_id + 1, w * dl_outputs.y);
        atomicAdd(temp + base_id + 2, w * dl_outputs.z);
        atomicAdd(temp + base_id + 3, w * dl_outputs.w);
    }
}
